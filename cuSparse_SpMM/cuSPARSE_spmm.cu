#include "cuSPARSE_spmm.h"

// template <typename Value>
void cuSparse_SPMM<float>::Preprocess(int m,int k,int nonzeros,
                    int *row_offsets,int* column_indices,float* values) {
        
    m_ = m; k_ = k;
    status= hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        return ;
    }
    hipsparseCreateCsr(&matA, m, k, nonzeros,
                    row_offsets, column_indices, values,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
}

// template <>
void cuSparse_SPMM<double>::Preprocess(int m,int k,int nonzeros,
                    int *row_offsets,int* column_indices,double* values){

    m_ = m; k_ = k;
    status= hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        return ;
    }
    hipsparseCreateCsr(&matA, m, k, nonzeros,
                    row_offsets, column_indices, values,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
}

// template <typename Value>
void cuSparse_SPMM<float>::Process(int n,float * B,float * C) {
    void *dBuffer = NULL;
    size_t bufferSize = 0;

    // Create dense matrix B
    int ldb = n;
    int ldc = n;

    hipsparseCreateDnMat(&matB, k_, n, ldb, B,
                                       HIP_R_32F, HIPSPARSE_ORDER_ROW);
    // Create dense matrix C
    hipsparseCreateDnMat(&matC, m_, n, ldc, C,
                                       HIP_R_32F, HIPSPARSE_ORDER_ROW);
    // allocate an external buffer if needed
    hipsparseSpMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);


    hipsparseSpMM(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);
}

// template <>
void cuSparse_SPMM<double>::Process(int n,double * B,double * C) {
    void *dBuffer = NULL;
    size_t bufferSize = 0;

    // Create dense matrix B
    int ldb = n;
    int ldc = n;

    hipsparseCreateDnMat(&matB, k_, n, ldb, B,
                                       HIP_R_64F, HIPSPARSE_ORDER_ROW);
    // Create dense matrix C
    hipsparseCreateDnMat(&matC, m_, n, ldc, C,
                                       HIP_R_64F, HIPSPARSE_ORDER_ROW);
    // allocate an external buffer if needed
    hipsparseSpMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_64F,
        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);


    hipsparseSpMM(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_64F,
                                HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);
}