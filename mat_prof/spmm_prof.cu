#include "hip/hip_runtime.h"
// for performance analysis

#include "hip/hip_runtime.h"
#include "matrix_utils.h"

#include "Sputnik_spmm.h"
#include "cuSPARSE_spmm.h"
#include "RoDeSpmm.h"

#include <sys/io.h>
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <dirent.h>
#include <stdlib.h>
#include <sys/stat.h>

using namespace std;
using namespace SPC;

#define SEG_LENGTH 512

#define BN 32

__global__ void MatrixDiff(int n,float* res,float* A,float* B) {
    if(threadIdx.x == 0 && blockIdx.x == 0)
        res[0] = 0.0f;
    
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= n) return;
    float diff = abs(A[idx] - B[idx]);

    // if(diff > 1e-5) {
    //     printf("[%d] : %f ~ %f\n",idx,A[idx],B[idx]);
    // }

    float r = diff;
    r += __shfl_down_sync(0xffffffff,r,16);
    r += __shfl_down_sync(0xffffffff,r,8);
    r += __shfl_down_sync(0xffffffff,r,4);
    r += __shfl_down_sync(0xffffffff,r,2);
    r += __shfl_down_sync(0xffffffff,r,1);

    if(threadIdx.x == 0)
        atomicAdd(res,r);

    __syncthreads();
    if(threadIdx.x == 0 && blockIdx.x == 0)
        printf("Matrix diff: %f\n",res[0]);
}

__global__ void PrintArray(int n,float* array) {
    for(int i=0; i < n; ++ i)
        printf("Array[%d]:%f\n",i,array[i]);
}

__global__ void PrintArrayInt(int n,int* array) {
    for(int i=0; i < n; ++ i)
        printf("IntArray[%d]:%d\n",i,array[i]);
}

int main(int argc,char **argv) {
    
    // hipSetDevice(0);

    // string file_path = "../../data/wv2010/wv2010.mtx";
    string file_path = "../../data/mip1/mip1.mtx";


    int ITER = 10;

    hipStream_t stream1,stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    double gflops = 0.0f;

    // cout<<file_path<<endl;

    SPC::SparseMatrix sm1(file_path,SPC::SORTED,1);

    int * row_offset_h = sm1.RowOffsets();
    int * row_indices_h = sm1.RowIndices();

    for(int i=0; i < 10; ++i)
        cout<<"  row["<<row_indices_h[i]<<"] : "<<row_offset_h[row_indices_h[i]+1] - row_offset_h[row_indices_h[i]]<<endl;


    sm1.RowDivide2Segment(SEG_LENGTH,4,32);
    
    SPC::CudaSparseMatrix<float> c_sm(sm1);

    int m = c_sm.Rows(), k = c_sm.Columns(), n = BN;

    absl::BitGen bitgen;
    SPC::CudaMatrix<float> d_B(k, n ,&bitgen);
    
    float* d_C;
    hipMalloc((void**)&d_C,sizeof(float)*m*n);

    float* d_C1;
    hipMalloc((void**)&d_C1,sizeof(float)*m*n);

    float* d_C2;
    hipMalloc((void**)&d_C2,sizeof(float)*m*n);

    float* diff;
    hipMalloc((void**)&diff,sizeof(float)*1);

    float tot_ms;
    hipEvent_t event1,event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    hipDeviceSynchronize();
    hipEventRecord(event1,0);
    for(int i=0; i < ITER; ++i)
        SPC::SputnikSpmm(m,c_sm.Columns(),n,c_sm.Nonzeros(),
                    c_sm.RowIndices(),c_sm.Values(),c_sm.RowOffsets(),c_sm.ColumnIndices(),
                    d_B.Values(),
                    d_C1,
                    stream1);

    hipEventRecord(event2,0);

    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&tot_ms, event1, event2);
    hipDeviceSynchronize();

    gflops = (double)ITER * (double)c_sm.Nonzeros() * 2 * n / tot_ms / 1000000;

    printf("Sputnik: %f, %f\n",tot_ms,gflops);


    cuSparse_SPMM<float> cu_sp;

    cu_sp.Preprocess(m,c_sm.Columns(),c_sm.Nonzeros(),
                    c_sm.RowOffsets(),c_sm.ColumnIndices(),c_sm.Values());

    hipDeviceSynchronize();
    hipEventRecord(event1,0);

    for(int i=0; i < ITER; ++i)
        cu_sp.Process(n,d_B.Values(),d_C);

    hipEventRecord(event2,0);

    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&tot_ms, event1, event2);
    hipDeviceSynchronize();

    gflops = (double)ITER * (double)c_sm.Nonzeros() * 2 * n / tot_ms / 1000000;

    printf("cuSparse: %f, %f\n",tot_ms,gflops);


    hipDeviceSynchronize();
    hipEventRecord(event1,0);

    for(int i = 0; i < ITER; ++i)
        RoDeSpmm_n32(c_sm.n_segs,c_sm.n_segs_residue,c_sm.Columns(),n,
                        c_sm.Values(),c_sm.ColumnIndices(),c_sm.RowOffsets(),
                       c_sm.seg_row_indices,c_sm.seg_row_indices_residue,c_sm.seg_st_offsets,
                       d_B.Values(),d_C2,stream1,stream2);

    hipEventRecord(event2,0);

    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&tot_ms, event1, event2);
    hipDeviceSynchronize();

    gflops = (double)ITER * (double)c_sm.Nonzeros() * 2 * n / tot_ms / 1000000;

    printf("Ours: %f, %f\n",tot_ms,gflops);

    // //    To validate, let ‘ITER’ be 1
    // MatrixDiff<<<(m*n+31)/32,32>>>(m*n,diff,d_C,d_C1);
    // MatrixDiff<<<(m*n+31)/32,32>>>(m*n,diff,d_C,d_C2);

    hipFree(d_C);
    hipFree(d_C1);
    hipFree(d_C2);
    hipFree(diff);
	
    return 0;
}
// 
