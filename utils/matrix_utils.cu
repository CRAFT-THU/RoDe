#include "hip/hip_runtime.h"
// Copyright 2020 The Sputnik Authors.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cstdint>
#include <cstring>
#include <numeric>

#include "glog/logging.h"
#include "matrix_utils.h"


#define METHOD_V1

namespace SPC {

namespace {

/**
 * @brief Helper to convert float data to half precision data.
 */
__global__ void ConvertKernel(const float *in_f, half2 *out, int n) {
  const float2 *in = reinterpret_cast<const float2 *>(in_f);
  n /= 2;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  out[idx] = __float22half2_rn(in[idx]);
}

__global__ void ConvertKernel(const int *in_i, short2 *out, int n) {
  const int2 *in = reinterpret_cast<const int2 *>(in_i);
  n /= 2;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  int2 a = in[idx];
  short2 b;
  b.x = static_cast<short>(a.x);
  b.y = static_cast<short>(a.y);
  out[idx] = b;
}

__global__ void ConvertKernel(const half2 *in, float *out_f, int n) {
  float2 *out = reinterpret_cast<float2 *>(out_f);
  n /= 2;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  out[idx] = __half22float2(in[idx]);
}

__global__ void ConvertKernel(const short2 *in, int *out_i, int n) {
  int2 *out = reinterpret_cast<int2 *>(out_i);
  n /= 2;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  short2 a = in[idx];
  int2 b;
  b.x = static_cast<int>(a.x);
  b.y = static_cast<int>(a.y);
  out[idx] = b;
}

__global__ void ConvertKernel(const float *in, double *out_i, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  out_i[idx] = (double) in[idx];
}

__global__ void ConvertKernel(const double *in, float *out_i, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  out_i[idx] = (float) in[idx];
}

/**
 * @brief Create a dense matrix with randomly sampled values.
 *
 * @param rows The number of rows in the matrix.
 * @param columns The number of columns in the matrix.
 * @param Buffer allocated to store the dense matirx.
 */
template <typename ValueType>
void MakeDenseMatrix(int rows, int columns, ValueType *matrix,
                     absl::BitGen *generator) {
  // Generate random values for the matrix.
  for (int64_t i = 0; i < static_cast<int64_t>(rows) * columns; ++i) {
    matrix[i] = absl::Uniform<ValueType>(*generator, -1, 1);
  }
}

void PadSparseMatrix(const std::vector<int> &row_offsets,
                     const std::vector<float> &values,
                     const std::vector<int> &column_indices, int row_padding,
                     std::vector<int> *row_offsets_out,
                     std::vector<float> *values_out,
                     std::vector<int> *column_indices_out) {
  CHECK_GE(row_padding, 0) << "Row padding factor must be greater than zero.";
  if (row_padding < 2) {
    // For row padding to the nearest 1 element, copy the input to the
    // output and return early. We also execute this code path for
    // `row_padding` == 0, which indicates no padding is to be added.
    row_offsets_out->assign(row_offsets.begin(), row_offsets.end());
    values_out->assign(values.begin(), values.end());
    column_indices_out->assign(column_indices.begin(), column_indices.end());
    return;
  }
  row_offsets_out->push_back(0);

  int offset = 0;
  for (int i = 0; i < row_offsets.size() - 1; ++i) {
    // Copy the existing values and column indices for this row to
    // the output.
    int row_length = row_offsets[i + 1] - row_offsets[i];
    values_out->resize(values_out->size() + row_length);
    column_indices_out->resize(column_indices_out->size() + row_length);
    std::copy(values.begin() + row_offsets[i],
              values.begin() + row_offsets[i + 1],
              values_out->begin() + offset);
    std::copy(column_indices.begin() + row_offsets[i],
              column_indices.begin() + row_offsets[i + 1],
              column_indices_out->begin() + offset);
    offset += row_length;

    // Calculate the number of zeros that need to be inserted in
    // this row to reach the desired padding factor.
    int residue = offset % row_padding;
    int to_add = (row_padding - residue) % row_padding;
    for (; to_add > 0; --to_add) {
      values_out->push_back(0.0);

      // NOTE: When we pad with zeros the column index that we assign
      // the phantom zero needs to be a valid column index s.t. we
      // don't index out-of-range into the dense rhs matrix when
      // computing spmm. Here we set all padding column-offsets to
      // the same column as the final non-padding weight in the row.
      column_indices_out->push_back(column_indices_out->back());
      ++offset;
    }
    row_offsets_out->push_back(offset);
  }
}

}  // namespace

template <typename In, typename Out>
hipError_t Convert(const In *in, Out *out, int n) {
  if (n == 0) return hipSuccess;
  CHECK_EQ(n % 2, 0) << "Number of elements must be multiple of 2.";

  int threads_per_block = 64;
  int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
  ConvertKernel<<<blocks_per_grid, threads_per_block, 0, 0>>>(in, out, n);
  return hipGetLastError();
}

template<>
hipError_t Convert(const float *in, float *out, int n) {
  return hipMemcpy(out, in, n * sizeof(float), hipMemcpyDeviceToDevice);
}

template<>
hipError_t Convert(const int *in, int *out, int n) {
  return hipMemcpy(out, in, n * sizeof(int), hipMemcpyDeviceToDevice);
}

template<>
hipError_t Convert(const float *in, double *out, int n) {
  if (n == 0) return hipSuccess;

  int threads_per_block = 64;
  int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
  ConvertKernel<<<blocks_per_grid, threads_per_block, 0, 0>>>(in, out, n);
  return hipGetLastError();
}

/**
 * @brief Create a sparse matrix with uniformly sampled non-zeros.
 *
 * @param rows The number of rows in the matrix.
 * @param columns The number of columns in the matrix.
 * @param nonzeros The number of non-zero values in the sparse matrix.
 * @param values Host-side buffer for the sparse matrix values.
 * Size `nonzeros + (row_padding - 1) * rows`.
 * @param row_offsets Host-side buffer for the sparse matrix row offsets.
 * Size `rows + 1`.
 * @param column_indices Host-side buffer for the sparse matrix column
 * offsets.
 * Size `nonzeros + (row_padding - 1) * rows`.
 * @param row_padding Each row in the sparse matrix will be padded to a
 * multiple of this value. Defaults to 4, which enables the user of
 * 4-element vector loads and stores. For best performance, pad to
 * `kBlockItemsK`.
 */
template <typename ValueType, typename IndexType>
void MakeSparseMatrixRandomUniform(int rows, int columns, int nonzeros,
                                   ValueType *values, IndexType *row_offsets,
                                   IndexType *column_indices,
                                   absl::BitGen *generator,
                                   int row_padding) {
  // The number of elements in the dense version of the matrix.
  int64_t num_elements = static_cast<int64_t>(rows) * columns;

  CHECK_LE(nonzeros, num_elements) << "The number of non-zero elements "
                                   << "must be <= the number of elements.";
  CHECK_GT(nonzeros, 0)
      << "The sparse matrix must have at least 1 non-zero value.";
  CHECK_GE(row_padding, 0) << "Row padding factor must be greater than zero.";

  // Generate random values for the matrix.
  std::vector<ValueType> nonzero_values(nonzeros);
  for (auto &v : nonzero_values) {
    v = absl::Uniform<ValueType>(*generator, -1, 1);
  }

  // Create a uniformly distributed random sparsity mask. We randomly
  // select which values to make zero and then mask them to create the
  // sparse matrix.
  std::vector<int64_t> indices(num_elements);
  std::iota(indices.begin(), indices.end(), 0);
  std::shuffle(indices.begin(), indices.end(), *generator);

  // Create the compressed sparse row indices and offsets.
  int64_t offset = 0;
  row_offsets[0] = 0;
  for (int64_t i = 0; i < rows; ++i) {
    for (int64_t j = 0; j < columns; ++j) {
      int64_t idx = i * columns + j;
      if (indices[idx] < nonzeros) {
        values[offset] = nonzero_values[indices[idx]];
        column_indices[offset] = j;
        ++offset;
      }
    }

    // If row_padding is zero, skip this code s.t. we don't mod zero.
    if (row_padding > 0) {
      // Pad the row with zeros s.t. every row contains a multiple of
      // `row_padding` elements.
      int residue = (offset - row_offsets[i]) % row_padding;
      int to_add = (row_padding - residue) % row_padding;
      for (; to_add > 0; --to_add) {
        values[offset] = 0.0;

        // NOTE: When we pad with zeros the column index that we assign
        // the phantom zero needs to be a valid column index s.t. we
        // don't index out-of-range into the dense rhs matrix when
        // computing spmm. Here we set all padding column-offsets to
        // the same column as the final non-padding weight in the row.
        column_indices[offset] = column_indices[offset - 1];
        ++offset;
      }

      // Set the row offset and sanity check the offset to make sure
      // we padded the row correctly.
      CHECK_EQ((offset - row_offsets[i]) % row_padding, 0);
    }
    row_offsets[i + 1] = offset;
  }
}

/**
 * @brief Create a sparse matrix with uniformly sampled non-zeros.
 *
 * @param rows The number of rows in the matrix.
 * @param columnss The number of columns in the matrix.
 * @param nonzeros_per_row The number of non-zero values in each row of
 * the sparse matrix.
 * @param values Host-side buffer for the sparse matrix values.
 * Size `nonzeros_per_row * rows`.
 * @param row_offsets Host-side buffer for the sparse matrix row offsets.
 * Size `rows + 1`.
 * @param column_indices Host-side buffer for the sparse matrix column
 * offsets.
 * Size `nonzeros_per_row * rows`.
 */
template <typename ValueType, typename IndexType>
void MakeSparseMatrixPerfectUniform(int rows, int columns, int nonzeros_per_row,
                                    ValueType *values, IndexType *row_offsets,
                                    IndexType *column_indices,
                                    absl::BitGen *generator) {
  // Generate random values for the matrix.
  int nonzeros = nonzeros_per_row * rows;
  for (int64_t i = 0; i < nonzeros; ++i) {
    values[i] = absl::Uniform<ValueType>(*generator, -1, 1);
  }

  // Select indices to make zero.
  int64_t offset = 0;
  std::vector<int64_t> indices(columns);
  std::iota(indices.begin(), indices.end(), 0);
  for (int64_t i = 0; i < rows; ++i) {
    std::shuffle(indices.begin(), indices.end(), *generator);
    std::vector<int64_t> sorted(nonzeros_per_row, 0);
    for (int64_t j = 0; j < nonzeros_per_row; ++j) {
      sorted[j] = indices[j];
    }
    std::sort(sorted.begin(), sorted.end());
    for (int64_t j = 0; j < nonzeros_per_row; ++j) {
      column_indices[offset + j] = sorted[j];
    }

    offset += nonzeros_per_row;
  }

  // Create the row offsets.
  offset = 0;
  for (int64_t i = 0; i < rows + 1; ++i) {
    row_offsets[i] = offset;
    offset += nonzeros_per_row;
  }
}

void IdentityRowSwizzle(int rows, const int * /* unused */, int *row_indices) {
  std::iota(row_indices, row_indices + rows, 0);
}

void SortedRowSwizzle(int rows, const int *row_offsets, int *row_indices) {
  // Create our unsorted row indices.
  std::vector<int> swizzle_staging(rows);
  std::iota(swizzle_staging.begin(), swizzle_staging.end(), 0);

  // Argsort the row indices based on their length.
  std::sort(swizzle_staging.begin(), swizzle_staging.end(),
            [&row_offsets](int idx_a, int idx_b) {
              int length_a = row_offsets[idx_a + 1] - row_offsets[idx_a];
              int length_b = row_offsets[idx_b + 1] - row_offsets[idx_b];
              return length_a > length_b;
            });

  // Copy the ordered row indices to the output.
  std::memcpy(row_indices, swizzle_staging.data(), sizeof(int) * rows);
}

void LoadBalanceSort(int rows, const int *row_offsets, int *row_indices,int BLOCK) {
  std::vector<int> swizzle_staging(rows);
  std::iota(swizzle_staging.begin(), swizzle_staging.end(), 0);
  
  int head_idx = 0, tail_idx = rows - 1, r_idx = 0;
  while(head_idx < rows) {
    swizzle_staging[head_idx++] = row_indices[r_idx ++];

    for(int bidx = 1; bidx < BLOCK; ++bidx){
      if(head_idx >= rows) break;
      swizzle_staging[head_idx++] = row_indices[tail_idx--];
    }
  }
  std::memcpy(row_indices, swizzle_staging.data(), sizeof(int) * rows);
}

SparseMatrix::SparseMatrix(int rows, int columns, int nonzeros,
                           ElementDistribution weight_distribution,
                           absl::BitGen *generator, Swizzle row_swizzle,
                           int pad_rows_to) {
  // Save the matrix meta-data.
  rows_ = rows;
  columns_ = columns;
  nonzeros_ = nonzeros;
  weight_distribution_ = weight_distribution;
  row_swizzle_ = row_swizzle;
  pad_rows_to_ = pad_rows_to;

  CHECK_LE(pad_rows_to_, columns)
      << "Rows cannot be padded to more values than there are columns.";

  // Create some temporary host-side buffers to build the matrix in.
  // Note that we have to pad these buffers to account for potential
  // extra storage requirements for row padding.
  int padding_elements = std::max((pad_rows_to_ - 1) * rows_, 0);
  std::vector<float> values_staging(nonzeros_ + padding_elements);
  std::vector<int> row_offsets_staging(rows_ + 1);
  std::vector<int> column_indices_staging(nonzeros_ + padding_elements);

  if (weight_distribution == RANDOM_UNIFORM) {
    MakeSparseMatrixRandomUniform(
        rows_, columns_, nonzeros_, values_staging.data(),
        row_offsets_staging.data(), column_indices_staging.data(), generator,
        pad_rows_to_);
  } else {
    // Verify that the number of nonzeros divides evenly into the
    // number of rows.
    CHECK_EQ(nonzeros_ % rows_, 0) << "The number of nonzeros must divide "
                                   << "evenly by the number of rows to "
                                   << "construct a PERFECT_UNIFORM matrix.";

    MakeSparseMatrixPerfectUniform(
        rows_, columns_, nonzeros_ / rows_, values_staging.data(),
        row_offsets_staging.data(), column_indices_staging.data(), generator);
  }

  // Figure out exactly how much storage we need for the padded matrices,
  // allocate the storage, and copy the matrices into our storage.
  num_elements_with_padding_ = row_offsets_staging[rows_];

  values_ = new float[num_elements_with_padding_];
  column_indices_ = new int[num_elements_with_padding_];
  row_offsets_ = new int[rows_ + 1];

  // Copy the data into our allocated buffers.
  std::memcpy(values_, values_staging.data(),
              num_elements_with_padding_ * sizeof(float));
  std::memcpy(column_indices_, column_indices_staging.data(),
              num_elements_with_padding_ * sizeof(int));
  std::memcpy(row_offsets_, row_offsets_staging.data(),
              (rows_ + 1) * sizeof(int));

  // Allocate storage for our swizzled row indices and set the values.
  row_indices_ = new int[rows_];
  if (row_swizzle_ == IDENTITY) {
    IdentityRowSwizzle(rows_, row_offsets_, row_indices_);
  } else {
    SortedRowSwizzle(rows_, row_offsets_, row_indices_);
  }

  // row_indices1 = new int[rows_];
  // row_indices2 = new int[rows_];
}

void SparseMatrix::InterLD(int BLOCK) {
  SortedRowSwizzle(rows_, row_offsets_, row_indices_);
  LoadBalanceSort(rows_,row_offsets_,row_indices_,BLOCK);
}

void SparseMatrix::InterLD_part(int BLOCK) {
  std::vector<int> swizzle_staging(nr1);

  int head_idx = 0, tail_idx = nr1 - 1, r_idx = 0;
  while(head_idx < nr1) {
    swizzle_staging[head_idx++] = row_indices_1[r_idx++];

    for(int bidx=1; bidx < BLOCK; ++ bidx) {
      if(head_idx >= nr1) break;
      swizzle_staging[head_idx++] = row_indices_1[tail_idx--];
    }
  }

  std::memcpy(row_indices_1,swizzle_staging.data(),sizeof(int)*nr1);
}

SparseMatrix::SparseMatrix(const CudaSparseMatrix<float> &sparse_matrix) {
  InitFromCudaSparseMatrix(sparse_matrix);
}

SparseMatrix::SparseMatrix(int rows, int columns, int nonzeros,
                           const std::vector<int> &row_offsets,
                           const std::vector<int> &column_indices,
                           absl::BitGen *generator, Swizzle row_swizzle,
                           int pad_rows_to)
    : rows_(rows),
      columns_(columns),
      nonzeros_(nonzeros),
      pad_rows_to_(pad_rows_to),
      weight_distribution_(RANDOM_UNIFORM),
      row_swizzle_(row_swizzle) {
  CHECK_LE(pad_rows_to_, columns)
      << "Rows cannot be padded to more values than there are columns.";

  // Generate random values for the sparse matrix parameters.
  std::vector<float> values(nonzeros_);
  for (auto &v : values) v = absl::Uniform<float>(*generator, -1, 1);

  // Pad the rows to the desired length.
  std::vector<int> row_offsets_staging, column_indices_staging;
  std::vector<float> values_staging;
  PadSparseMatrix(row_offsets, values, column_indices, pad_rows_to,
                  &row_offsets_staging, &values_staging,
                  &column_indices_staging);

  // Figure out exactly how much storage we need for the padded matrices,
  // allocate the storage, and copy the matrices into our storage.
  num_elements_with_padding_ = row_offsets_staging[rows_];

  values_ = new float[num_elements_with_padding_];
  column_indices_ = new int[num_elements_with_padding_];
  row_offsets_ = new int[rows_ + 1];

  // Copy the data into our allocated buffers.
  std::memcpy(values_, values_staging.data(),
              num_elements_with_padding_ * sizeof(float));
  std::memcpy(column_indices_, column_indices_staging.data(),
              num_elements_with_padding_ * sizeof(int));
  std::memcpy(row_offsets_, row_offsets_staging.data(),
              (rows_ + 1) * sizeof(int));

  // Allocate storage for our swizzled row indices and set the values.
  row_indices_ = new int[rows_];
  if (row_swizzle_ == IDENTITY) {
    IdentityRowSwizzle(rows_, row_offsets_, row_indices_);
  } else {
    SortedRowSwizzle(rows_, row_offsets_, row_indices_);
  }
}

void SparseMatrix::InitFromCudaSparseMatrix(
    const CudaSparseMatrix<float> &sparse_matrix) {
  // Copy the sparse matrix meta-data.
  rows_ = sparse_matrix.Rows();
  columns_ = sparse_matrix.Columns();
  nonzeros_ = sparse_matrix.Nonzeros();
  pad_rows_to_ = sparse_matrix.PadRowsTo();
  num_elements_with_padding_ = sparse_matrix.NumElementsWithPadding();
  weight_distribution_ = sparse_matrix.WeightDistribution();
  row_swizzle_ = sparse_matrix.RowSwizzle();

  // Allocate memory on the CPU for our matrix.
  values_ = new float[num_elements_with_padding_];
  column_indices_ = new int[num_elements_with_padding_];
  row_offsets_ = new int[rows_ + 1];
  row_indices_ = new int[rows_];

  // Copy the results to the CPU.
  CUDA_CALL(hipMemcpy(values_, sparse_matrix.Values(),
                       sizeof(float) * num_elements_with_padding_,
                       hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(column_indices_, sparse_matrix.ColumnIndices(),
                       sizeof(int) * num_elements_with_padding_,
                       hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(row_offsets_, sparse_matrix.RowOffsets(),
                       sizeof(int) * (rows_ + 1), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(row_indices_, sparse_matrix.RowIndices(),
                       sizeof(int) * rows_, hipMemcpyDeviceToHost));
}

template <typename Value>
CudaSparseMatrix<Value>::CudaSparseMatrix(
    int rows, int columns, int nonzeros,
    ElementDistribution weight_distribution, absl::BitGen *generator,
    Swizzle row_swizzle, int pad_rows_to) {
  CHECK_EQ(pad_rows_to % TypeUtils<Value>::kElementsPerScalar, 0)
      << "The number of elements in each row must be divisible by "
      << "the number of elements per scalar value for the specified "
      << "data type.";
  // Create a sparse matrix on the host.
  SparseMatrix sparse_matrix(rows, columns, nonzeros, weight_distribution,
                             generator, row_swizzle, pad_rows_to);
  InitFromSparseMatrix(sparse_matrix);
}

template <typename Value>
CudaSparseMatrix<Value>::CudaSparseMatrix(const SparseMatrix &sparse_matrix) {
  // The number of nonzeros in each row must be divisible by the number of
  // elements per scalar for the specified data type.
  for (int i = 0; i < sparse_matrix.Rows(); ++i) {
    int nnz = sparse_matrix.RowOffsets()[i + 1] - sparse_matrix.RowOffsets()[i];
    CHECK_EQ(nnz % TypeUtils<Value>::kElementsPerScalar, 0)
        << "The number of elements in each row must be divisible by "
        << "the number of elements per scalar value for the specified "
        << "data type.";
  }
  InitFromSparseMatrix(sparse_matrix);
}

template <typename Value>
void CudaSparseMatrix<Value>::InitFromSparseMatrix(
    const SparseMatrix &sparse_matrix) {
  // Copy the sparse matrix meta-data.
  rows_ = sparse_matrix.Rows();
  columns_ = sparse_matrix.Columns();
  nonzeros_ = sparse_matrix.Nonzeros();
  pad_rows_to_ = sparse_matrix.PadRowsTo();
  num_elements_with_padding_ = sparse_matrix.NumElementsWithPadding();
  weight_distribution_ = sparse_matrix.WeightDistribution();
  row_swizzle_ = sparse_matrix.RowSwizzle();

  // Allocate memory on the GPU for our matrix.
  float *values_float = nullptr;
  int *column_indices_int = nullptr;
  CUDA_CALL(
      hipMalloc(&values_float, sizeof(float) * num_elements_with_padding_));
  CUDA_CALL(hipMalloc(&column_indices_int,
                       sizeof(int) * num_elements_with_padding_));
  CUDA_CALL(hipMalloc(&row_offsets_, sizeof(int) * (rows_ + 1)));
  CUDA_CALL(hipMalloc(&row_indices_, sizeof(int) * rows_));

  // Copy the results to the GPU.
  CUDA_CALL(hipMemcpy(values_float, sparse_matrix.Values(),
                       sizeof(float) * num_elements_with_padding_,
                       hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(column_indices_int, sparse_matrix.ColumnIndices(),
                       sizeof(int) * num_elements_with_padding_,
                       hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(row_offsets_, sparse_matrix.RowOffsets(),
                       sizeof(int) * (rows_ + 1), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(row_indices_, sparse_matrix.RowIndices(),
                       sizeof(int) * rows_, hipMemcpyHostToDevice));
  CUDA_CALL(hipStreamSynchronize(nullptr));

  // Allocate memory for the values and indices in the target datatype.
  int elements =
      num_elements_with_padding_ / TypeUtils<Value>::kElementsPerScalar;
  CUDA_CALL(hipMalloc(&values_, sizeof(Value) * elements));
  CUDA_CALL(hipMalloc(&column_indices_, sizeof(Index) * elements));

  // Convert to the target datatype.
  CUDA_CALL(Convert(values_float, values_, num_elements_with_padding_));
  CUDA_CALL(
      Convert(column_indices_int, column_indices_, num_elements_with_padding_));
  CUDA_CALL(hipStreamSynchronize(nullptr));

  // Free the temporary memory.
  CUDA_CALL(hipFree(values_float));
  CUDA_CALL(hipFree(column_indices_int));


  nr1 = sparse_matrix.nr1;
  nr2 = sparse_matrix.nr2;

  row_indices_1 = nullptr;
  row_indices_2 = nullptr;
  if(nr1 > 0){
    CUDA_CALL(hipMalloc((void**)&row_indices_1,sizeof(int)*nr1));
    CUDA_CALL(hipMemcpy(row_indices_1,sparse_matrix.row_indices_1,sizeof(int)*nr1,hipMemcpyHostToDevice));
  }
  if(nr2 > 0) {
    CUDA_CALL(hipMalloc((void**)&row_indices_2,sizeof(int)*nr2));
    CUDA_CALL(hipMemcpy(row_indices_2,sparse_matrix.row_indices_2,sizeof(int)*nr2,hipMemcpyHostToDevice));
  }

  seg_row_indices = nullptr;
  seg_st_offsets = nullptr;
  seg_row_indices_residue = nullptr;

  n_segs = sparse_matrix.n_segs;
  
  if(n_segs > 0) {
    CUDA_CALL(hipMalloc((void**)&seg_row_indices,sizeof(int)*n_segs));
    CUDA_CALL(hipMemcpy(seg_row_indices,sparse_matrix.seg_row_indices,sizeof(int)*n_segs,hipMemcpyHostToDevice));
  
    CUDA_CALL(hipMalloc((void**)&seg_st_offsets,sizeof(int)*(n_segs+1)));
    CUDA_CALL(hipMemcpy(seg_st_offsets,sparse_matrix.seg_st_offsets,sizeof(int)*(n_segs+1),hipMemcpyHostToDevice));
  }

  n_segs_residue = sparse_matrix.n_segs_residue;
  if(n_segs_residue > 0) {
    CUDA_CALL(hipMalloc((void**)&seg_row_indices_residue,sizeof(int)*n_segs_residue));
    CUDA_CALL(hipMemcpy(seg_row_indices_residue,sparse_matrix.seg_row_indices_residue,sizeof(int)*n_segs_residue,hipMemcpyHostToDevice));
  }
  
}

Matrix::Matrix(int rows, int columns, absl::BitGen *generator) {
  rows_ = rows;
  columns_ = columns;

  // Allocate storage for the matrix
  values_ = new float[rows_ * columns_];
  MakeDenseMatrix(rows_, columns_, values_, generator);
}

template <typename Value>
void Matrix::InitFromCudaMatrix(const CudaMatrix<Value> &matrix) {
  // Copy the matrix meta-data.
  rows_ = matrix.Rows();
  columns_ = matrix.Columns();

  // Allocate memory for our matrix.
  values_ = new float[rows_ * columns_];

  // Allocate a temporary buffer on GPU to convert the values into.
  float *matrix_values_float = nullptr;
  CUDA_CALL(hipMalloc(&matrix_values_float, sizeof(float) * rows_ * columns_));
  CUDA_CALL(Convert(matrix.Values(), matrix_values_float, rows_ * columns_));

  // Copy the results from the GPU.
  CUDA_CALL(hipMemcpy(values_, matrix_values_float,
                       sizeof(float) * rows_ * columns_,
                       hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(matrix_values_float));
}

template <typename Value>
CudaMatrix<Value>::CudaMatrix(int rows, int columns, absl::BitGen *generator) {
  // Create a dense matrix on the host and copy to gpu.
  Matrix matrix(rows, columns, generator);
  InitFromMatrix(matrix);
}

template <typename Value>
CudaMatrix<Value>::CudaMatrix(const Matrix &matrix) {
  InitFromMatrix(matrix);
}

template <typename Value>
void CudaMatrix<Value>::InitFromMatrix(const Matrix &matrix) {
  // Copy the matrix meta-data.
  rows_ = matrix.Rows();
  columns_ = matrix.Columns();

  // Allocate memory on the GPU for our matrix.
  float *values_float = nullptr;
  CUDA_CALL(hipMalloc(&values_float, sizeof(float) * rows_ * columns_));

  // Copy the results to the GPU.
  CUDA_CALL(hipMemcpy(values_float, matrix.Values(),
                       sizeof(float) * rows_ * columns_,
                       hipMemcpyHostToDevice));
  CUDA_CALL(hipStreamSynchronize(nullptr));

  // Allocate memory for the values in the target data type.
  size_t elements = rows_ * columns_ / TypeUtils<Value>::kElementsPerScalar;
  CUDA_CALL(hipMalloc(&values_, sizeof(Value) * elements));

  // Convert to the target data type.
  CUDA_CALL(Convert(values_float, values_, rows_ * columns_));
  CUDA_CALL(hipStreamSynchronize(nullptr));

  // Free the temporary memory.
  CUDA_CALL(hipFree(values_float));
}

// Explicit instantiations for template functions and classes.
template class CudaSparseMatrix<float>;
template class CudaSparseMatrix<half2>;
template class CudaSparseMatrix<double>;

template class CudaMatrix<float>;
template class CudaMatrix<half2>;
template class CudaMatrix<double>;

template void Matrix::InitFromCudaMatrix(const CudaMatrix<float> &);
template void Matrix::InitFromCudaMatrix(const CudaMatrix<half2> &);
template void Matrix::InitFromCudaMatrix(const CudaMatrix<double> &);


int compare0(const void *a, const void *b)
{
  if (((struct v_struct *)a)->row - ((struct v_struct *)b)->row > 0) return 1;
  if (((struct v_struct *)a)->row - ((struct v_struct *)b)->row < 0) return -1;
  if (((struct v_struct *)a)->col - ((struct v_struct *)b)->col > 0) return 1;
  if (((struct v_struct *)a)->col - ((struct v_struct *)b)->col < 0) return -1;
  return 0;
}

SparseMatrix::SparseMatrix(const std::string& file_path, Swizzle row_swizzle,int pad_rows_to) {

  // printf("begin reading...\n");

  srand(0);

  row_swizzle_ = row_swizzle;
  pad_rows_to_ = pad_rows_to;

  char buf[300];
  int nflag, sflag;

  FILE *fp = fopen(file_path.c_str(),"r");
  fgets(buf, 300, fp);
  
  if(strstr(buf, "symmetric") != NULL || strstr(buf, "Hermitian") != NULL) sflag = 1; // symmetric
  else sflag = 0;

  if(strstr(buf, "pattern") != NULL) nflag = 0; // non-value
  else if(strstr(buf, "complex") != NULL) nflag = -1;
  else nflag = 1;

  #ifdef SYM
      sflag = 1;
  #endif

  int pre_count = 0;
  while(1) {
      pre_count++;
      fgets(buf, 300, fp);
      if(strstr(buf, "%") == NULL) break;
  }
  fclose(fp);

  int i;
  fp = fopen(file_path.c_str(), "r");
  for(i=0;i<pre_count;i++)
    fgets(buf, 300, fp);


	fscanf(fp, "%d %d %d", &rows_, &columns_, &nonzeros_);

  // printf("row: %d column:%d nnz: %d\n",rows_,columns_,nonzeros_);
	nonzeros_ *= (sflag+1);

	struct v_struct *temp_v = (struct v_struct *)malloc(sizeof(struct v_struct)*(nonzeros_+1));
	// gold_temp_v = (struct v_struct *)malloc(sizeof(struct v_struct)*(nonzeros_+1));
	for(i=0;i<nonzeros_;i++) {

		fscanf(fp, "%d %d", &temp_v[i].row, &temp_v[i].col);
		// temp_v[i].grp = INIT_GRP;
		temp_v[i].row--; temp_v[i].col--;

		if(temp_v[i].row < 0 || temp_v[i].row >= rows_ || temp_v[i].col < 0 || temp_v[i].col >= columns_) {
			fprintf(stdout, "A vertex id is out of range %d %d\n", temp_v[i].row, temp_v[i].col);
			exit(0);
		}
		if(nflag == 0) temp_v[i].val = (float)(rand()%1048576)/1048576;
		else if(nflag == 1) {
				float ftemp;
				fscanf(fp, " %f ", &ftemp);
				temp_v[i].val = ftemp;
		} else { // complex
				float ftemp1, ftemp2;
				fscanf(fp, " %f %f ", &ftemp1, &ftemp2);
				temp_v[i].val = ftemp1;
		}
    // ??
		// temp_v[i].val = (float)(rand()%1048576)/1048576;
		#ifdef SIM_VALUE
		  temp_v[i].val = 1.0f;
		#endif

		if(sflag == 1) {
			i++;
			temp_v[i].row = temp_v[i-1].col;
			temp_v[i].col = temp_v[i-1].row;
			temp_v[i].val = temp_v[i-1].val;
			// temp_v[i].grp = INIT_GRP;
		}
	}
  fclose(fp);
	qsort(temp_v, nonzeros_, sizeof(struct v_struct), compare0);

  // remove repeated-elements
	int *loc = (int *)malloc(sizeof(int)*(nonzeros_+1));
	memset(loc, 0, sizeof(int)*(nonzeros_+1));
	loc[0]=1;
	for(i=1;i<nonzeros_;i++) {
		if(temp_v[i].row == temp_v[i-1].row && temp_v[i].col == temp_v[i-1].col)
			loc[i] = 0;
		else loc[i] = 1;
	}
	for(i=1;i<=nonzeros_;i++)
    loc[i] += loc[i-1];
	for(i=nonzeros_; i>=1; i--)
    loc[i] = loc[i-1];
	loc[0] = 0;
	for(i=0;i<nonzeros_;i++) {
		temp_v[loc[i]].row = temp_v[i].row;
		temp_v[loc[i]].col = temp_v[i].col;
		temp_v[loc[i]].val = temp_v[i].val;
		// temp_v[loc[i]].grp = temp_v[i].grp;
	}
	nonzeros_ = loc[nonzeros_];
	temp_v[nonzeros_].row = rows_;
	free(loc);

  std::vector<float> values_staging(nonzeros_);
  std::vector<int> row_offsets_staging(rows_ + 1);
  std::vector<int> column_indices_staging(nonzeros_);

  // std::iota(row_offsets_staging.begin(), row_offsets_staging.end(), 0);
  std::fill(row_offsets_staging.begin(),row_offsets_staging.end(),0);
	
	for(i=0; i< nonzeros_ ;i++) {
		column_indices_staging[i] = temp_v[i].col;
		values_staging[i] = temp_v[i].val;
		row_offsets_staging[1+temp_v[i].row] = i+1;
	}

	for(i=1;i< rows_ ;i++) {
    if(row_offsets_staging[i] == 0) row_offsets_staging[i] = row_offsets_staging[i-1];
	}
	row_offsets_staging[rows_] = nonzeros_;
  free(temp_v);

  std::vector<int> row_offsets_staging1, column_indices_staging1;
  std::vector<float> values_staging1;
  PadSparseMatrix(row_offsets_staging, values_staging, column_indices_staging, pad_rows_to,
                  &row_offsets_staging1, &values_staging1,
                  &column_indices_staging1);

  num_elements_with_padding_ = row_offsets_staging1[rows_];

  values_ = new float[num_elements_with_padding_];
  column_indices_ = new int[num_elements_with_padding_];
  row_offsets_ = new int[rows_ + 1];

  std::memcpy(values_, values_staging1.data(),
              num_elements_with_padding_ * sizeof(float));
  std::memcpy(column_indices_, column_indices_staging1.data(),
              num_elements_with_padding_ * sizeof(int));
  std::memcpy(row_offsets_, row_offsets_staging1.data(),
              (rows_ + 1) * sizeof(int));

  row_indices_ = new int[rows_];
  if (row_swizzle_ == IDENTITY) {
    IdentityRowSwizzle(rows_, row_offsets_, row_indices_);
  } else {
    SortedRowSwizzle(rows_, row_offsets_, row_indices_);
  }

  row_indices_1 = new int[rows_];
  row_indices_2 = new int[rows_];

  seg_row_indices = nullptr;
  seg_st_offsets  = nullptr;
  n_segs = 0;

  seg_row_indices_residue = nullptr;
  n_segs_residue = 0;
}

template <typename T>
__global__ void square_sum(int n,const T* v1,const T* v2,T* res) {
  T r = 0.0f;
  for(int i = threadIdx.x; i < n; i += blockDim.x) {
    r += (v1[i] - v2[i]) * (v1[i] - v2[i]);
  }

  r += __shfl_down_sync(0xffffffff,r,16);
  r += __shfl_down_sync(0xffffffff,r,8);
  r += __shfl_down_sync(0xffffffff,r,4);
  r += __shfl_down_sync(0xffffffff,r,2);
  r += __shfl_down_sync(0xffffffff,r,1);

  if((threadIdx.x & 31) == 0) 
    atomicAdd(&res[0],r);

}

template <typename T>
void Matrix_Diff(const CudaMatrix<T>& matrix1,const CudaMatrix<T>& matrix2) {
  int n = matrix1.Rows() * matrix1.Columns();

  T * res, *_res;

  hipMalloc((void**)&_res,sizeof(T));
  res = new T[1];

  res[0] = 0.0f;
  hipMemcpy(_res,res,sizeof(T),hipMemcpyHostToDevice);

  square_sum<T><<<1,1024>>>(n,matrix1.Values(),matrix2.Values(),_res);

  hipMemcpy(res,_res,sizeof(T),hipMemcpyDeviceToHost);

  hipFree(_res);

  printf("Diff : %f",res[0] / n);

  delete[] res;
  return;
}

#ifdef METHOD_V1
  void SparseMatrix::RowDivide(int vectorLen,int K) {
    nr1 = 0;
    nr2 = 0;

    for(int i=0; i < rows_; ++ i) {
      int r_idx = row_indices_[i];
      int n_padding = row_offsets_[r_idx] % vectorLen;
      int nnz = row_offsets_[r_idx + 1] - row_offsets_[r_idx] + n_padding;
      if(nnz >= K) {
        row_indices_1[nr1++] = row_indices_[i];
      }
      if(nnz % K) {
        row_indices_2[nr2++] = row_indices_[i];
      }

    }

  }
#else
  void SparseMatrix::RowDivide(int vectorLen,int K) {
    nr1 = 0;
    nr2 = 0;

    for(int i=0; i < rows_; ++ i) {
      int r_idx = row_indices_[i];
      int n_padding = row_offsets_[r_idx] % vectorLen;
      int nnz = row_offsets_[r_idx + 1] - row_offsets_[r_idx] + n_padding;
      if(nnz >= K) {
        row_indices_1[nr1++] = row_indices_[i];
      }
      else {
        row_indices_2[nr2++] = row_indices_[i];
      }

    }
  }
#endif

void SparseMatrix::RowDivide2Segment(int SegmentLength,int vectorLen,int KBLOCK) {
  int nr = 0;
  std::vector<int> row_indices_staging;
  std::vector<int> st_offsets_staging;
  std::vector<int> row_indices_residue_staging;
  
  for(int i=0; i < rows_; ++i) {
    int row_offset = row_offsets_[i];
    int n_padding = row_offset % vectorLen;
    int nnz = row_offsets_[i+1] - row_offset + n_padding;

    if(nnz > SegmentLength) {
      row_indices_staging.push_back(i);
      st_offsets_staging.push_back(row_offset);
      row_offset = (row_offset + SegmentLength) - n_padding;

      nnz -= SegmentLength;
    }

    while(nnz > SegmentLength) {
      row_indices_staging.push_back(i);
      st_offsets_staging.push_back(row_offset);

      row_offset += SegmentLength;
      nnz -= SegmentLength;
    }

    if(nnz > 0) {
      if(nnz >= KBLOCK){
        row_indices_staging.push_back(i);
        st_offsets_staging.push_back(row_offset);
      }
      if( nnz % KBLOCK) {
        row_indices_residue_staging.push_back(i);
      }
    }
  }

  st_offsets_staging.push_back(row_offsets_[rows_]);

  if(n_segs > 0) {
    delete[] seg_row_indices;
    delete[] seg_st_offsets;
  }
  if(n_segs_residue > 0) {
    delete[] seg_row_indices_residue;
  }

  n_segs = row_indices_staging.size();
  seg_row_indices = new int[n_segs];
  seg_st_offsets = new int[n_segs+1];

  n_segs_residue = row_indices_residue_staging.size();
  seg_row_indices_residue = new int[n_segs_residue];

  std::memcpy(seg_row_indices,row_indices_staging.data(),sizeof(int)*n_segs);
  std::memcpy(seg_st_offsets,st_offsets_staging.data(),sizeof(int)*(n_segs+1));
  std::memcpy(seg_row_indices_residue,row_indices_residue_staging.data(),sizeof(int)*n_segs_residue);
}

void SparseMatrix::RowDivide2Segment_Nopadding(int SegmentLength,int vectorLen,int KBLOCK) {
  int nr = 0;
  std::vector<int> row_indices_staging;
  std::vector<int> st_offsets_staging;
  std::vector<int> row_indices_residue_staging;
  
  for(int i=0; i < rows_; ++i) {
    int row_offset = row_offsets_[i];
    int nnz = row_offsets_[i+1] - row_offset ;

    while(nnz > SegmentLength) {
      row_indices_staging.push_back(i);
      st_offsets_staging.push_back(row_offset);

      row_offset += SegmentLength;
      nnz -= SegmentLength;
    }

    if(nnz > 0) {
      if(nnz >= KBLOCK){
        row_indices_staging.push_back(i);
        st_offsets_staging.push_back(row_offset);
      }
      if( nnz % KBLOCK) {
        row_indices_residue_staging.push_back(i);
      }
    }
  }

  st_offsets_staging.push_back(row_offsets_[rows_]);

  n_segs = row_indices_staging.size();
  seg_row_indices = new int[n_segs];
  seg_st_offsets = new int[n_segs+1];

  n_segs_residue = row_indices_residue_staging.size();
  seg_row_indices_residue = new int[n_segs_residue];

  std::memcpy(seg_row_indices,row_indices_staging.data(),sizeof(int)*n_segs);
  std::memcpy(seg_st_offsets,st_offsets_staging.data(),sizeof(int)*(n_segs+1));
  std::memcpy(seg_row_indices_residue,row_indices_residue_staging.data(),sizeof(int)*n_segs_residue);
}

template void Matrix_Diff(const CudaMatrix<float>& matrix1,const CudaMatrix<float>& matrix2);

}  


